
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void square(float * d_out, float * d_in) {
	int idx = threadIdx.x;
	float f = d_in[idx];
	d_out[idx] = f * f;
}

int main(int argc, char** argv){
	const int ARRAY_SIZE = 64;
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

	// Generate the input array on the host
	float h_in[ARRAY_SIZE];
	for (int i = 0; i < ARRAY_SIZE; i++){
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// Declare GPU memory pointers
	float * d_in;
	float * d_out;

	// Allocate GPU memory
	hipMalloc((void **) &d_in, ARRAY_BYTES);
	hipMalloc((void **) &d_out, ARRAY_BYTES);

	// Transfert the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// Launch kernel
	square<<<1, ARRAY_SIZE>>>(d_out, d_in);

	// Copy back the result array to the CPU
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	for (int i = 0; i < ARRAY_SIZE; i++){
		printf("%f", h_out[i]);
		printf((i % 4) != 3 ? "\t" : "\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}
